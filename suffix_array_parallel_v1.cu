#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include <string.h>

//__global__ void strcmp(int* result, char* str, int i, int n) {
//	// Strings to be compared
//	int x = blockIdx.x, y = threadIdx.x;
//
//	int xy = x * n + y;
//
//	// If start of comparison
//	if (i == 0)
//		result[xy] = str[x] - str[y];
//	
//	// Previous result is zero, i.e. undecided
//	else if (result[xy] == 0) {
//		int i1 = x + i, i2 = y + i;
//
//		// Check if within bounds
//		if (i1 < n && i2 < n)
//			result[xy] = str[i1] - str[i2];
//	}
//}

__global__ void strcmp(int* result, char* str, int n) {
	// Strings to be compared
	int x = blockIdx.x, y = threadIdx.x;

	//int xy = x * n + y;

	// If start of comparison
	int r = str[x] - str[y], i;

	for (i = 1; i < n; i++) {
		if (r != 0) {
			break;
		}

		int i1 = x + i, i2 = y + i;
		if (i1 < n && i2 < n) {
			r = str[i1] - str[i2];
		}
		else break;
	}
	result[x * n + y] = r;
}

__global__ void oddeven(int* result, int* arr, int odd, int n) {
	int id = blockIdx.x;
	int lower = id * 2 + odd;
	int higher = lower + 1;

	// Check if within bounds
	if (higher < n) {
		int xy = arr[lower] * n + arr[higher];
		//printf("\nid=%d odd=%d low=%d high=%d compIndex=%d compVal=%d\n", id, odd, lower, higher, xy, result[xy]);

		//printf("Before: %d\t%d\t%d\t%d\t%d\n", arr[0], arr[1], arr[2], arr[3], arr[4]);
		// If string comparison is negative then swap
		if (result[xy] > 0) {
			int temp = arr[lower];
			arr[lower] = arr[higher];
			arr[higher] = temp;
		}

		//printf("After: %d\t%d\t%d\t%d\t%d\n", arr[0], arr[1], arr[2], arr[3], arr[4]);
	}
}

void randomDnaCodeGenerator(int size, char string[]) {
	int i;
	char a = 'A';
	char c = 'C';
	char g = 'G';
	char t = 'T';
	char dollar = '$';
	//time_t tick;

	//srand((unsigned) time(&tick));

	for (i = 0; i < size-1; i++) {
		switch (rand() % 4) {
		case 0: string[i] = a;
			break;
		case 1: string[i] = c;
			break;
		case 2: string[i] = g;
			break;
		case 3: string[i] = t;
			break;
		}
	}

	string[size] = dollar;
	string[size + 1] = 0;
}

int main() {
	int size = 10000;
	char dna[10000];

	randomDnaCodeGenerator(size, dna);

	printf("DNA: %s\n\n", dna);

	int arr[sizeof(dna) / sizeof(char)];

	int n = 0;
	while (dna[n] != '$')
	{
		arr[n] = n;
		n++;
	}
	arr[n] = n; // For '$'

	n++; // n is the number of Elements

	char* cudaDNA;
	int * cudaARR;

	int rSize = n * n;
	int* cudaResult;

	float elapsed = 0;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&cudaDNA, n * sizeof(char));
	hipMemcpy(cudaDNA, dna, n * sizeof(char), hipMemcpyHostToDevice);

	hipMalloc(&cudaARR, n * sizeof(int));
	hipMemcpy(cudaARR, arr, n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&cudaResult, rSize * sizeof(int));

	int i;

	hipEventRecord(start);
	//for (i = 0; i < n; i++)
		//strcmp << < n, n >> >(cudaResult, cudaDNA, i, n);

		strcmp << < n, n >> >(cudaResult, cudaDNA, n);

	for(i = 0;i < n; i++)
		oddeven <<<n / 2, 2 >>>(cudaResult, cudaARR, i % 2, n);
	hipEventRecord(stop);

	int* result;
	result = (int*) malloc(rSize * sizeof(int));
	hipMemcpy(result, cudaResult, rSize * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(arr, cudaARR, n * sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);

	/*for (i = 1; i <= rSize; i++) {
		printf("%d ", result[i-1]);
		if (i % n == 0)
			printf("\n");
	}*/

	//printf("\n");
	/*for (i = 0; i < n; i++)
		printf("%d ", arr[i]);*/

	printf("\n\nElapsed Time: %f", elapsed);

	hipFree(cudaDNA);
	hipFree(cudaARR);
	hipFree(cudaResult);
	free(result);

	getch();
	return 0;
}